#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

//
// Matrix multiplication: C = A * B.
// Host code.
//
// This sample implements matrix multiplication as described in Chapter 3
// of the programming guide and uses the CUBLAS library to demonstrate
// the best performance.

// SOME PRECAUTIONS:
// IF WE WANT TO CALCULATE ROW-MAJOR MATRIX MULTIPLY C = A * B,
// WE JUST NEED CALL CUBLAS API IN A REVERSE ORDER: cublasSegemm(B, A)! 
// The reason is explained as follows:

// CUBLAS library uses column-major storage, but C/C++ use row-major storage.
// When passing the matrix pointer to CUBLAS, the memory layout alters from 
// row-major to column-major, which is equivalent to an implict transpose. 

// In the case of row-major C/C++ matrix A, B, and a simple matrix multiplication 
// C = A * B, we can't use the input order like hipblasSgemm(A, B)  because of
// implict transpose. The actual result of cublasSegemm(A, B) is A(T) * B(T). 
// If col(A(T)) != row(B(T)), equal to row(A) != col(B), A(T) and B(T) are not 
// multipliable. Moreover, even if A(T) and B(T) are multipliable, the result C 
// is a column-based cublas matrix, which means C(T) in C/C++, we need extra 
// transpose code to convert it to a row-based C/C++ matrix.

// To solve the problem, let's consider our desired result C, a row-major matrix. 
// In cublas format, it is C(T) actually (becuase of the implict transpose). 
// C = A * B, so C(T) = (A * B) (T) = B(T) * A(T). Cublas matrice B(T) and A(T) 
// happen to be C/C++ matrice B and A (still becuase of the implict transpose)! 
// We don't need extra transpose code, we only need alter the input order!
//
// CUBLAS provides high-performance matrix multiplication.
// See also:
// V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
// in Proc. 2008 ACM/IEEE Conf. on Superconducting (SC '08),
// Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
//

#include "matrixMul.cuh"

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set matrix multiply on CPU
//! C = A * B
//! @param C          reference data, computed but preallocated
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param hA         height of matrix A
//! @param wB         width of matrix B
////////////////////////////////////////////////////////////////////////////////
void
matrixMulCPU(float *C, const float *A, const float *B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j)
        {
            double sum = 0;

            for (unsigned int k = 0; k < wA; ++k)
            {
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }

            C[i * wB + j] = (float)sum;
        }
}

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions (in addition to hip/hip_runtime_api.h)

void inline checkError(hipblasStatus_t status, const char *msg)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("%s", msg);
        exit(EXIT_FAILURE);
    }
}
// end of CUDA Helper Functions

// Allocates a matrix with random float entries.
void randomInit(float *data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height, int iListLength, float fListTol)
{
    printf("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
    int i,j,k;
    int error_count=0;

    for (j = 0; j < height; j++)
    {
        if (error_count < iListLength)
        {
            printf("\n  Row %d:\n", j);
        }

        for (i = 0; i < width; i++)
        {
            k = j * width + i;
            float fDiff = fabs(data1[k] - data2[k]);

            if (fDiff > fListTol)
            {
                if (error_count < iListLength)
                {
                    printf("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j, data1[k], data2[k], fDiff);
                }

                error_count++;
            }
        }
    }

    printf(" \n  Total Errors = %d\n", error_count);
}

void initializeCUDA(int &devID, int &iSizeMultiple, sMatrixSize &matrix_size)
{
    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    hipError_t error;
    devID = 0;

    error = hipSetDevice(devID);

    if (error != hipSuccess)
    {
        printf("hipSetDevice returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }


    // get number of SMs on this GPU
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    iSizeMultiple = min(iSizeMultiple, 10);
    iSizeMultiple = max(iSizeMultiple, 1);

    hipDeviceProp_t deviceProp;

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

    // use a larger block size for Fermi and above
    int block_size = (deviceProp.major < 2) ? 16 : 32;

    matrix_size.uiWA = 2 * block_size * iSizeMultiple;
    matrix_size.uiHA = 4 * block_size * iSizeMultiple;
    matrix_size.uiWB = 2 * block_size * iSizeMultiple;
    matrix_size.uiHB = 4 * block_size * iSizeMultiple;
    matrix_size.uiWC = 2 * block_size * iSizeMultiple;
    matrix_size.uiHC = 4 * block_size * iSizeMultiple;

    printf("MatrixA(%u,%u), MatrixB(%u,%u), MatrixC(%u,%u)\n",
           matrix_size.uiWA, matrix_size.uiHA,
           matrix_size.uiWB, matrix_size.uiHB,
           matrix_size.uiWC, matrix_size.uiHC);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test matrix multiply using CUBLAS
////////////////////////////////////////////////////////////////////////////////
int matrixMultiply(int devID, sMatrixSize &matrix_size)
{
    hipDeviceProp_t deviceProp;
    hipError_t error;

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // use a larger block size for Fermi and above
    int block_size = (deviceProp.major < 2) ? 16 : 32;

    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = matrix_size.uiWA * matrix_size.uiHA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = matrix_size.uiWB * matrix_size.uiHB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);

    // set seed for rand()
    srand(2006);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    unsigned int size_C = matrix_size.uiWC * matrix_size.uiHC;
    unsigned int mem_size_C = sizeof(float) * size_C;

    // allocate host memory for the result
    float *h_C      = (float *) malloc(mem_size_C);
    float *h_CUBLAS = (float *) malloc(mem_size_C);

    error = hipMalloc((void **) &d_A, mem_size_A);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_B, mem_size_B);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_B returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // copy host memory to device
    error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy d_A h_A returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy d_B h_B returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_C, mem_size_C);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_C returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(matrix_size.uiWC / threads.x, matrix_size.uiHC / threads.y);

    // create and start timer
    printf("Computing result using CUBLAS...");

    // execute the kernel
    int nIter = 30;

    // CUBLAS version 2.0
    {
        hipblasHandle_t handle;

        hipblasStatus_t ret;

        ret = hipblasCreate(&handle);

        if (ret != HIPBLAS_STATUS_SUCCESS)
        {
            printf("hipblasCreate returned error code %d, line(%d)\n", ret, __LINE__);
            exit(EXIT_FAILURE);
        }

        const float alpha = 1.0f;
        const float beta  = 0.0f;
        //Perform warmup operation with cublas
        ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiWB, matrix_size.uiHA, matrix_size.uiWA, &alpha, d_B, matrix_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWA);

        if (ret != HIPBLAS_STATUS_SUCCESS)
        {
            printf("hipblasSgemm returned error code %d, line(%d)\n", ret, __LINE__);
            exit(EXIT_FAILURE);
        }

        // Allocate CUDA events that we'll use for timing
        hipEvent_t start;
        error = hipEventCreate(&start);

        if (error != hipSuccess)
        {
            fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
            exit(EXIT_FAILURE);
        }

        hipEvent_t stop;
        error = hipEventCreate(&stop);

        if (error != hipSuccess)
        {
            fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
            exit(EXIT_FAILURE);
        }

        // Record the start event
        error = hipEventRecord(start, NULL);

        if (error != hipSuccess)
        {
            fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
            exit(EXIT_FAILURE);
        }

        for (int j = 0; j < nIter; j++)
        {
            //note cublas is column primary!
            //need to transpose the order
            ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiWB, matrix_size.uiHA, matrix_size.uiWA, &alpha, d_B, matrix_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWA);

            if (ret != HIPBLAS_STATUS_SUCCESS)
            {
                printf("hipblasSgemm returned error code %d, line(%d)\n", ret, __LINE__);
                exit(EXIT_FAILURE);
            }
        }

        printf("done.\n");

        // Record the stop event
        error = hipEventRecord(stop, NULL);

        if (error != hipSuccess)
        {
            fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
            exit(EXIT_FAILURE);
        }

        // Wait for the stop event to complete
        error = hipEventSynchronize(stop);

        if (error != hipSuccess)
        {
            fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
            exit(EXIT_FAILURE);
        }

        float msecTotal = 0.0f;
        error = hipEventElapsedTime(&msecTotal, start, stop);

        if (error != hipSuccess)
        {
            fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
            exit(EXIT_FAILURE);
        }

        // Compute and print the performance
        float msecPerMatrixMul = msecTotal / nIter;
        double flopsPerMatrixMul = 2.0 * (double)matrix_size.uiWA * (double)matrix_size.uiHA * (double)matrix_size.uiWB;
        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
        printf(
            "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",
            gigaFlops,
            msecPerMatrixMul,
            flopsPerMatrixMul);

        // copy result from device to host
        error = hipMemcpy(h_CUBLAS, d_C, mem_size_C, hipMemcpyDeviceToHost);

        if (error != hipSuccess)
        {
            printf("hipMemcpy h_CUBLAS d_C returned error code %d, line(%d)\n", error, __LINE__);
            exit(EXIT_FAILURE);
        }

        checkError(hipblasDestroy(handle), "hipblasDestroy() error!\n");
    }

    // compute reference solution
    printf("Computing result using host CPU...");
    float *reference = (float *)malloc(mem_size_C);
    
	clock_t ini = clock();
	//for(int iVuelta = 0; iVuelta < nIter; iVuelta++)
	matrixMulCPU(reference, h_A, h_B, matrix_size.uiHA, matrix_size.uiWA, matrix_size.uiWB);

	clock_t fin = clock();

    printf("done.\n");

    // check result (CUBLAS)
    bool resCUBLAS = sdkCompareL2fe(reference, h_CUBLAS, size_C, 1.0e-6f);

    if (resCUBLAS != true)
    {
        printDiff(reference, h_CUBLAS, matrix_size.uiWC, matrix_size.uiHC, 100, 1.0e-5f);
    }

	printf("Tiempo en CPU: %fms\n", ((double)(fin-ini))/CLOCKS_PER_SEC *1000.0); fflush(0);

    printf("Comparing CUBLAS Matrix Multiply with CPU results: %s\n", (true == resCUBLAS) ? "PASS" : "FAIL");

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset();

    if (resCUBLAS == true)
    {
        return EXIT_SUCCESS;    // return value = 1
    }
    else
    {
        return EXIT_FAILURE;     // return value = 0
    }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int lanzarProceso()
{
    printf("[Matrix Multiply CUBLAS] - Starting...\n");

    int devID = 0, sizeMult = 10;
    sMatrixSize matrix_size;

    initializeCUDA(devID, sizeMult, matrix_size);

    int matrix_result = matrixMultiply(devID, matrix_size);

	return matrix_result;
    //exit(matrix_result);
}
